#include "hip/hip_runtime.h"
#define BLOCK_SIZE 32

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <iostream>
#include <time.h>
#include <hipblas.h>
#include <hip/hip_cooperative_groups.h>

#include "QCA_constants.cuh"
#include "QCA_parameters.cuh"
#include "support.cuh"
#include "naiveSolver.cuh"
#include "parallelSolver1.cuh"
#include "kernel.cuh"
#include "util.cuh"
#include "hip/hip_runtime.h"

void stepCalculation_p_v1(matrix & rho, butcher & butcher_DP45, double step, lindbladOperators & lindblad, matrix & H)
{
	hipError_t cudaReturn;

	hipDoubleComplex *rho_d, *rhoPlus1_d, *rhoPlus1_ec_d, *k1_d, *k2_d, *k3_d, *k4_d, *k5_d, *k6_d, *k7_d;
	hipDoubleComplex *k1_prepare_d, *k2_prepare_d, *k3_prepare_d, *k4_prepare_d, *k5_prepare_d, *k6_prepare_d, *k7_prepare_d;
	hipDoubleComplex *lindblad1_d, *lindblad2_d, *H_d;

	size_t size_all = rhoSize * rhoSize;

	hipMalloc((void**)&rho_d, sizeof(hipDoubleComplex) * size_all);
	hipMalloc((void**)&rhoPlus1_d, sizeof(hipDoubleComplex) * size_all);
	hipMalloc((void**)&rhoPlus1_ec_d, sizeof(hipDoubleComplex) * size_all);

	hipMalloc((void**)&k1_d, sizeof(hipDoubleComplex) * size_all);
	hipMalloc((void**)&k2_d, sizeof(hipDoubleComplex) * size_all);
	hipMalloc((void**)&k3_d, sizeof(hipDoubleComplex) * size_all);
	hipMalloc((void**)&k4_d, sizeof(hipDoubleComplex) * size_all);
	hipMalloc((void**)&k5_d, sizeof(hipDoubleComplex) * size_all);
	hipMalloc((void**)&k6_d, sizeof(hipDoubleComplex) * size_all);
	hipMalloc((void**)&k7_d, sizeof(hipDoubleComplex) * size_all);

	hipMalloc((void**)&k1_prepare_d, sizeof(hipDoubleComplex) * size_all);
	hipMalloc((void**)&k2_prepare_d, sizeof(hipDoubleComplex) * size_all);
	hipMalloc((void**)&k3_prepare_d, sizeof(hipDoubleComplex) * size_all);
	hipMalloc((void**)&k4_prepare_d, sizeof(hipDoubleComplex) * size_all);
	hipMalloc((void**)&k5_prepare_d, sizeof(hipDoubleComplex) * size_all);
	hipMalloc((void**)&k6_prepare_d, sizeof(hipDoubleComplex) * size_all);
	hipMalloc((void**)&k7_prepare_d, sizeof(hipDoubleComplex) * size_all);

	hipMalloc((void**)&lindblad1_d, sizeof(hipDoubleComplex) * size_all);
	hipMalloc((void**)&lindblad2_d, sizeof(hipDoubleComplex) * size_all);
	hipMalloc((void**)&H_d, sizeof(hipDoubleComplex) * size_all);

	hipMemcpy(rho_d, rho.elements, sizeof(hipDoubleComplex) * size_all, hipMemcpyHostToDevice);
	hipMemcpy(rhoPlus1_d, rho.elements, sizeof(hipDoubleComplex) * size_all, hipMemcpyHostToDevice);
	hipMemcpy(rhoPlus1_ec_d, rho.elements, sizeof(hipDoubleComplex) * size_all, hipMemcpyHostToDevice);

	hipMemcpy(lindblad1_d, lindblad.lindblad_1_m.elements, sizeof(hipDoubleComplex) * size_all, hipMemcpyHostToDevice);
	hipMemcpy(lindblad2_d, lindblad.lindblad_2_m.elements, sizeof(hipDoubleComplex) * size_all, hipMemcpyHostToDevice);
	hipMemcpy(H_d, H.elements, sizeof(hipDoubleComplex) * size_all, hipMemcpyHostToDevice);

	hipDeviceSynchronize();

	clock_t timerStart;
	clock_t timerStop;
	double timer;

	timerStart = clock();

	// blockDim, gridDim, ==> launch kernel
	dim3 blockDim3(BLOCK_SIZE, BLOCK_SIZE);
	dim3 gridDim3((rhoSize + BLOCK_SIZE - 1) / BLOCK_SIZE, (rhoSize + BLOCK_SIZE - 1) / BLOCK_SIZE);

	matrixInit_cuda << <blockDim3, gridDim3 >> > (k1_d);
	matrixInit_cuda << <blockDim3, gridDim3 >> > (k1_prepare_d);
	RHS_cuda(k1_d, rho_d, H_d, lindblad1_d, lindblad2_d);
	matrixScale_cuda << <blockDim3, gridDim3 >> > (k1_d, make_hipDoubleComplex(step, 0));

	matrixInit_cuda << <blockDim3, gridDim3 >> > (k2_d);
	matrixInit_cuda << <blockDim3, gridDim3 >> > (k2_prepare_d);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k2_prepare_d, rho_d, make_hipDoubleComplex(1.0, 0.0));
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k2_prepare_d, k1_d, butcher_DP45.A1_m[0]);
	RHS_cuda(k2_d, k2_prepare_d, H_d, lindblad1_d, lindblad2_d);
	matrixScale_cuda << <blockDim3, gridDim3 >> > (k2_d, make_hipDoubleComplex(step, 0));

	matrixInit_cuda << <blockDim3, gridDim3 >> > (k3_d);
	matrixInit_cuda << <blockDim3, gridDim3 >> > (k3_prepare_d);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k3_prepare_d, rho_d, make_hipDoubleComplex(1.0, 0.0));
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k3_prepare_d, k1_d, butcher_DP45.A2_m[0]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k3_prepare_d, k2_d, butcher_DP45.A2_m[1]);
	RHS_cuda(k3_d, k3_prepare_d, H_d, lindblad1_d, lindblad2_d);
	matrixScale_cuda << <blockDim3, gridDim3 >> > (k3_d, make_hipDoubleComplex(step, 0));

	matrixInit_cuda << <blockDim3, gridDim3 >> > (k4_d);
	matrixInit_cuda << <blockDim3, gridDim3 >> > (k4_prepare_d);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k4_prepare_d, rho_d, make_hipDoubleComplex(1.0, 0.0));
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k4_prepare_d, k1_d, butcher_DP45.A3_m[0]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k4_prepare_d, k2_d, butcher_DP45.A3_m[1]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k4_prepare_d, k3_d, butcher_DP45.A3_m[2]);
	RHS_cuda(k4_d, k4_prepare_d, H_d, lindblad1_d, lindblad2_d);
	matrixScale_cuda << <blockDim3, gridDim3 >> > (k4_d, make_hipDoubleComplex(step, 0));

	matrixInit_cuda << <blockDim3, gridDim3 >> > (k5_d);
	matrixInit_cuda << <blockDim3, gridDim3 >> > (k5_prepare_d);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k5_prepare_d, rho_d, make_hipDoubleComplex(1.0, 0.0));
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k5_prepare_d, k1_d, butcher_DP45.A4_m[0]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k5_prepare_d, k2_d, butcher_DP45.A4_m[1]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k5_prepare_d, k3_d, butcher_DP45.A4_m[2]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k5_prepare_d, k4_d, butcher_DP45.A4_m[3]);
	RHS_cuda(k5_d, k5_prepare_d, H_d, lindblad1_d, lindblad2_d);
	matrixScale_cuda << <blockDim3, gridDim3 >> > (k5_d, make_hipDoubleComplex(step, 0));

	matrixInit_cuda << <blockDim3, gridDim3 >> > (k6_d);
	matrixInit_cuda << <blockDim3, gridDim3 >> > (k6_prepare_d);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k6_prepare_d, rho_d, make_hipDoubleComplex(1.0, 0.0));
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k6_prepare_d, k1_d, butcher_DP45.A5_m[0]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k6_prepare_d, k2_d, butcher_DP45.A5_m[1]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k6_prepare_d, k3_d, butcher_DP45.A5_m[2]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k6_prepare_d, k4_d, butcher_DP45.A5_m[3]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k6_prepare_d, k5_d, butcher_DP45.A5_m[4]);
	RHS_cuda(k6_d, k6_prepare_d, H_d, lindblad1_d, lindblad2_d);
	matrixScale_cuda << <blockDim3, gridDim3 >> > (k6_d, make_hipDoubleComplex(step, 0));

	matrixInit_cuda << <blockDim3, gridDim3 >> > (k7_d);
	matrixInit_cuda << <blockDim3, gridDim3 >> > (k7_prepare_d);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k7_prepare_d, rho_d, make_hipDoubleComplex(1.0, 0.0));
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k7_prepare_d, k1_d, butcher_DP45.A6_m[0]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k7_prepare_d, k2_d, butcher_DP45.A6_m[1]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k7_prepare_d, k3_d, butcher_DP45.A6_m[2]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k7_prepare_d, k4_d, butcher_DP45.A6_m[3]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k7_prepare_d, k5_d, butcher_DP45.A6_m[4]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (k7_prepare_d, k6_d, butcher_DP45.A6_m[5]);
	RHS_cuda(k7_d, k7_prepare_d, H_d, lindblad1_d, lindblad2_d);
	matrixScale_cuda << <blockDim3, gridDim3 >> > (k7_d, make_hipDoubleComplex(step, 0));

	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (rhoPlus1_d, k1_d, butcher_DP45.B1_m[0]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (rhoPlus1_d, k2_d, butcher_DP45.B1_m[1]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (rhoPlus1_d, k3_d, butcher_DP45.B1_m[2]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (rhoPlus1_d, k4_d, butcher_DP45.B1_m[3]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (rhoPlus1_d, k5_d, butcher_DP45.B1_m[4]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (rhoPlus1_d, k6_d, butcher_DP45.B1_m[5]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (rhoPlus1_d, k7_d, butcher_DP45.B1_m[6]);

	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (rhoPlus1_ec_d, k1_d, butcher_DP45.B2_m[0]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (rhoPlus1_ec_d, k2_d, butcher_DP45.B2_m[1]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (rhoPlus1_ec_d, k3_d, butcher_DP45.B2_m[2]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (rhoPlus1_ec_d, k4_d, butcher_DP45.B2_m[3]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (rhoPlus1_ec_d, k5_d, butcher_DP45.B2_m[4]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (rhoPlus1_ec_d, k6_d, butcher_DP45.B2_m[5]);
	matrixScaleAdd_cuda << <blockDim3, gridDim3 >> > (rhoPlus1_ec_d, k7_d, butcher_DP45.B2_m[6]);

	hipDeviceSynchronize();

	// free up memory
	hipFree(k1_d);
	hipFree(k2_d);
	hipFree(k3_d);
	hipFree(k4_d);
	hipFree(k5_d);
	hipFree(k6_d);
	hipFree(k7_d);

	hipFree(rho_d);
	hipFree(rhoPlus1_d);
	hipFree(rhoPlus1_ec_d);

	hipFree(k1_prepare_d);
	hipFree(k2_prepare_d);
	hipFree(k3_prepare_d);
	hipFree(k4_prepare_d);
	hipFree(k5_prepare_d);
	hipFree(k6_prepare_d);
	hipFree(k7_prepare_d);

	hipFree(H_d);
	hipFree(lindblad1_d);
	hipFree(lindblad2_d);

	std::cout << "step calculation finished" << std::endl;

	timerStop = clock();
	timer = (double)(timerStop - timerStart) / CLOCKS_PER_SEC;
	printf("time elapsed in this one step is: %f \n", timer);
}

void RHS_cuda(hipDoubleComplex * drhodt, hipDoubleComplex * x, hipDoubleComplex * H, hipDoubleComplex * lindblad1, hipDoubleComplex * lindblad2)
{
	dim3 blockDim3(BLOCK_SIZE, BLOCK_SIZE);
	dim3 gridDim3((rhoSize + BLOCK_SIZE - 1) / BLOCK_SIZE, (rhoSize + BLOCK_SIZE - 1) / BLOCK_SIZE);

	hipDoubleComplex * inter;
	hipMalloc((void**)&inter, sizeof(hipDoubleComplex) * rhoSize * rhoSize);
	matrixInit_cuda << <gridDim3, blockDim3 >> > (inter);

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipDoubleComplex alpha, beta;

	// lindblad1
	alpha = make_hipDoubleComplex(1, 0);
	beta = make_hipDoubleComplex(1, 0);

	hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rhoSize, rhoSize, rhoSize, &alpha, lindblad1, rhoSize, x, rhoSize, &beta, inter, rhoSize);
	hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_C, rhoSize, rhoSize, rhoSize, &alpha, inter, rhoSize, lindblad1, rhoSize, &beta, drhodt, rhoSize);

	matrixInit_cuda << <gridDim3, blockDim3 >> > (inter);

	hipblasZgemm(handle, HIPBLAS_OP_C, HIPBLAS_OP_N, rhoSize, rhoSize, rhoSize, &alpha, lindblad1, rhoSize, lindblad1, rhoSize, &beta, inter, rhoSize);

	alpha = make_hipDoubleComplex(-0.5, 0.0);
	hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rhoSize, rhoSize, rhoSize, &alpha, inter, rhoSize, x, rhoSize, &beta, drhodt, rhoSize);
	hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rhoSize, rhoSize, rhoSize, &alpha, x, rhoSize, inter, rhoSize, &beta, drhodt, rhoSize);

	// lindblad2
	alpha = make_hipDoubleComplex(1, 0);
	beta = make_hipDoubleComplex(1, 0);

	hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rhoSize, rhoSize, rhoSize, &alpha, lindblad2, rhoSize, x, rhoSize, &beta, inter, rhoSize);
	hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_C, rhoSize, rhoSize, rhoSize, &alpha, inter, rhoSize, lindblad2, rhoSize, &beta, drhodt, rhoSize);

	matrixInit_cuda << <gridDim3, blockDim3 >> > (inter);

	hipblasZgemm(handle, HIPBLAS_OP_C, HIPBLAS_OP_N, rhoSize, rhoSize, rhoSize, &alpha, lindblad2, rhoSize, lindblad2, rhoSize, &beta, inter, rhoSize);

	alpha = make_hipDoubleComplex(-0.5, 0.0);
	hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rhoSize, rhoSize, rhoSize, &alpha, inter, rhoSize, x, rhoSize, &beta, drhodt, rhoSize);
	hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rhoSize, rhoSize, rhoSize, &alpha, x, rhoSize, inter, rhoSize, &beta, drhodt, rhoSize);

	hipFree(inter);

	// commutator
	alpha = make_hipDoubleComplex(0, -1.0 / hbarEv);
	hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rhoSize, rhoSize, rhoSize, &alpha, H, rhoSize, x, rhoSize, &beta, drhodt, rhoSize);

	alpha = make_hipDoubleComplex(0, 1.0 / hbarEv);
	hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rhoSize, rhoSize, rhoSize, &alpha, x, rhoSize, H, rhoSize, &beta, drhodt, rhoSize);

	hipblasDestroy(handle);
	std::cout << "RHS calculation finished" << std::endl;
}
