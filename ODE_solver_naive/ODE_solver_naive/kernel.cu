#include "hip/hip_runtime.h"
#define BLOCK_SIZE 32

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <iostream>
#include <time.h>
#include <hipblas.h>

#include "QCA_constants.cuh"
#include "QCA_parameters.cuh"
#include "support.cuh"
#include "naiveSolver.cuh"
#include "parallelSolver1.cuh"
#include "kernel.cuh"
#include "util.cuh"
#include "hip/hip_runtime.h"
#include ""

__global__ void matrixInit_cuda(hipDoubleComplex * mat)
{
	unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < rhoSize && col < rhoSize)
	{
		mat[row * rhoSize + col] = make_hipDoubleComplex(0.0, 0.0);
	}
}

__global__ void matrixScaleAdd_cuda(hipDoubleComplex * A, hipDoubleComplex * B, hipDoubleComplex scaler)
{
	// A += B * scaler
	unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < rhoSize && col < rhoSize)
	{
		A[row * rhoSize + col] = hipCadd(A[row * rhoSize + col], hipCmul(B[row * rhoSize + col], scaler));
	}
}

__global__ void matrixScale_cuda(hipDoubleComplex * A, hipDoubleComplex scaler)
{
	unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < rhoSize && col < rhoSize)
	{
		A[row * rhoSize + col] = hipCmul(A[row * rhoSize + col], scaler);
	}
}

__global__ void matrixSub_cuda(hipDoubleComplex * A, hipDoubleComplex * B)
{
	unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < rhoSize && col < rhoSize)
	{
		A[row * rhoSize + col] = hipCsub(A[row * rhoSize + col], B[row * rhoSize + col]);
	}
}
